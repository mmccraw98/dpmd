#include "hip/hip_runtime.h"
#include "kernels/base_poly_particle_kernels.cuh"
#include "stdio.h"

namespace md::poly {

__constant__ PolyConst g_poly;
__constant__ PolySystemConst g_vertex_sys;

void bind_poly_globals(const int* d_particle_id, const int* d_particle_offset, const int* d_n_vertices_per_particle) {
    PolyConst h { d_particle_id, d_particle_offset, d_n_vertices_per_particle };
    hipMemcpyToSymbol(HIP_SYMBOL(g_poly), &h, sizeof(PolyConst));
}
void bind_poly_system_globals(const int* d_vertex_system_offset, const int* d_vertex_system_id, const int* d_vertex_system_size) {
    PolySystemConst h { d_vertex_system_offset, d_vertex_system_id, d_vertex_system_size };
    hipMemcpyToSymbol(HIP_SYMBOL(g_vertex_sys), &h, sizeof(PolySystemConst));
}

__global__ void count_naive_vertex_neighbors_kernel(
    int* __restrict__ neighbor_count
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int Nv = md::geo::g_sys.n_vertices;
    if (i >= Nv) return;

    const int v_sid = g_vertex_sys.id[i];
    const int v_sys_size = g_vertex_sys.size[v_sid];
    const int v_pid = g_poly.particle_id[i];
    const int n_vertices_in_particle = g_poly.n_vertices_per_particle[v_pid];
    // if there are N_v_t total vertices in the system, a vertex of particle i has a maximum of (N_v_t - N_v_i) vertex neighbors
    const int n_neighbors = v_sys_size - n_vertices_in_particle;
    neighbor_count[i] = n_neighbors;
}

__global__ void fill_naive_vertex_neighbor_list_kernel(
    const int* __restrict__ neighbor_start,
    int* __restrict__ neighbor_ids
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int Nv = md::geo::g_sys.n_vertices;
    if (i >= Nv) return;

    const int v_sid = g_vertex_sys.id[i];
    const int v_sys_beg = g_vertex_sys.offset[v_sid];
    const int v_sys_end = g_vertex_sys.offset[v_sid+1];
    const int v_pid = g_poly.particle_id[i];

    int neigh_pid;

    int w = neighbor_start[i];
    for (int j = v_sys_beg; j < v_sys_end; ++j) {
        neigh_pid = g_poly.particle_id[j];
        if (neigh_pid == v_pid) continue;
        neighbor_ids[w++] = j;
    }
}

}

