#include "hip/hip_runtime.h"
#include "particles/disk.cuh"
#include "integrators/velocity_verlet.cuh"
#include "integrators/damped_velocity_verlet.cuh"
#include <cmath>
#include <algorithm>
#include "utils/h5_io.hpp"

int main(int argc, char** argv) {
    if (argc != 6) {
        std::cerr << "Usage: " << argv[0] << " <num_particles_per_system> <dt_scale> <n_steps> <S> <out_path>" << std::endl;
        return 1;
    }
    const int num_particles_per_system = atoi(argv[1]);
    const double dt_scale = atof(argv[2]);
    const int n_steps = atoi(argv[3]);
    const int S = atoi(argv[4]);
    std::string out_path = argv[5];

    const int n_cell_dim = 4;
    const double packing_fraction = 0.3;
    const double rad = 0.5;
    const double mass = 1.0;
    const double e_interaction = 1.0;
    const int N = num_particles_per_system * S;
    const double box_size = std::sqrt(num_particles_per_system * M_PI * rad * rad / packing_fraction);

    std::vector<int> host_cell_size_dim(S);
    std::vector<int> host_system_size(S);
    std::vector<int> host_system_start(S + 1);
    std::vector<int> host_cell_system_start(S + 1);
    std::vector<double> host_rad(N);
    std::vector<double> host_mass(N);
    std::vector<double> host_e_interaction(S);
    std::vector<double> host_skin(S);
    std::vector<double> host_thresh2(S);
    std::vector<double> host_box_size(S);
    std::vector<int> host_system_id(N);
    std::vector<double> host_pos_x(N), host_pos_y(N), host_force_x(N), host_force_y(N), host_pe(N);
    std::vector<int> host_neighbor_ids;
    std::vector<int> host_neighbor_start;
    std::vector<double> host_area(N);
    host_system_start[0] = 0;
    host_cell_system_start[0] = 0;
    for (int i = 0; i < S; i++) {
        host_cell_size_dim[i] = n_cell_dim;
        host_system_size[i] = num_particles_per_system;
        host_system_start[i + 1] = host_system_start[i] + num_particles_per_system;
        host_cell_system_start[i + 1] = host_cell_system_start[i] + n_cell_dim * n_cell_dim;
        host_box_size[i] = box_size;
        host_e_interaction[i] = e_interaction;
        host_skin[i] = 2.0 * rad;
        host_thresh2[i] = (0.5 * host_skin[i]) * (0.5 * host_skin[i]);
        for (int j = 0; j < num_particles_per_system; j++) {
            host_system_id[host_system_start[i] + j] = i;
        }
    }
    for (int i = 0; i < N; i++) {
        host_mass[i] = mass;
        host_rad[i] = rad;
        host_area[i] = M_PI * rad * rad;
    }

    df::DeviceField1D<double> dt; dt.resize(S); dt.fill(dt_scale);


    md::disk::Disk P;
    P.set_neighbor_method(md::NeighborMethod::Naive); // set this before allocating particles

    P.allocate_systems(S);
    P.allocate_particles(N);

    P.neighbor_cutoff.from_host(host_skin);
    P.thresh2.from_host(host_thresh2);
    P.system_id.from_host(host_system_id);
    P.system_size.from_host(host_system_size);
    P.system_offset.from_host(host_system_start);
    P.cell_dim.from_host(host_cell_size_dim, host_cell_size_dim);
    P.cell_system_start.from_host(host_cell_system_start);
    P.box_size.from_host(host_box_size, host_box_size);
    P.e_interaction.from_host(host_e_interaction);
    P.rad.from_host(host_rad);
    P.mass.from_host(host_mass);
    P.rad.from_host(host_rad);
    P.pos.stateless_rand_uniform(0.0, box_size, 0.0, box_size, 0);
    P.area.from_host(host_area);

    P.sync_box();
    P.sync_system();
    P.sync_neighbors();
    P.sync_cells();
    P.sync_class_constants();
    P.init_neighbors();
    P.compute_packing_fraction();

    {  // Equilibrate initially
        df::DeviceField1D<double> damping(S); damping.fill(1.0);
        md::integrators::DampedVelocityVerlet dvv(P, dt, damping);
        dvv.init();
        for (int i = 0; i < n_steps; i++) {
            dvv.step();
        }
    }

    double vel_scale = 5e-2;
    P.vel.stateless_rand_uniform(-vel_scale, vel_scale, -vel_scale, vel_scale);

    md::integrators::VelocityVerlet vv(P, dt);
    vv.init();

    int n_saves = 0;
    hid_t out_file = H5Fcreate(out_path.c_str(), H5F_ACC_TRUNC, H5P_DEFAULT, H5P_DEFAULT);
    if (out_file < 0) { std::cerr << "Failed to create " << out_path << "\n"; return 1; }

    std::vector<double> out_pos_x, out_pos_y;
    std::vector<double> out_force_x, out_force_y;
    std::vector<double> out_packing_fraction;
    std::vector<double> out_box_size_x, out_box_size_y;
    std::vector<double> out_pe_total;
    std::vector<double> out_ke_total;
    std::vector<double> out_rad; P.rad.to_host(out_rad); write_vector(out_file, "rad", out_rad);
    std::vector<int> out_sys_offset; P.system_offset.to_host(out_sys_offset); write_vector(out_file, "system_offset", out_sys_offset);

    int save_increment = std::max(1, n_steps / 1000);

    for (int i = 0; i < n_steps; i++) {
        vv.step();
        if (i % save_increment == 0) {
            P.compute_packing_fraction();
            P.compute_pe_total();
            P.compute_ke_total();

            std::string group_name = "step_" + std::to_string(n_saves);
            hid_t group = H5Gcreate2(out_file, group_name.c_str(), H5P_DEFAULT, H5P_DEFAULT, H5P_DEFAULT);

            P.pos.to_host(out_pos_x, out_pos_y); write_vector_2d(group, "pos", out_pos_x, out_pos_y);
            P.force.to_host(out_force_x, out_force_y); write_vector_2d(group, "force", out_force_x, out_force_y);
            P.packing_fraction.to_host(out_packing_fraction); write_vector(group, "packing_fraction", out_packing_fraction);
            P.box_size.to_host(out_box_size_x, out_box_size_y); write_vector_2d(group, "box_size", out_box_size_x, out_box_size_y);
            P.pe_total.to_host(out_pe_total); write_vector(group, "pe_total", out_pe_total);
            P.ke_total.to_host(out_ke_total); write_vector(group, "ke_total", out_ke_total);

            n_saves++;
        }
    }

    write_scalar(out_file, "n_saves", n_saves);
    write_scalar(out_file, "n_particles", N);
    write_scalar(out_file, "n_systems", S);

    
    // std::vector<int> out_vertex_sys_offset; P.vertex_system_offset.to_host(out_vertex_sys_offset); write_vector(out_file, "vertex_system_offset", out_vertex_sys_offset);
    // std::vector<int> out_vertex_offset; P.particle_offset.to_host(out_vertex_offset); write_vector(out_file, "vertex_offset", out_vertex_offset);
    // std::vector<double> out_angle; P.angle.to_host(out_angle); write_vector(out_file, "angle", out_angle);
    // std::vector<double> out_vertex_pos_x, out_vertex_pos_y; P.vertex_pos.to_host(out_vertex_pos_x, out_vertex_pos_y); write_vector_2d(out_file, "vertex_pos", out_vertex_pos_x, out_vertex_pos_y);
    // std::vector<double> out_vertex_rad; P.vertex_rad.to_host(out_vertex_rad); write_vector(out_file, "vertex_rad", out_vertex_rad);
    // std::vector<double> out_vertex_force_x, out_vertex_force_y; P.vertex_force.to_host(out_vertex_force_x, out_vertex_force_y); write_vector_2d(out_file, "vertex_force", out_vertex_force_x, out_vertex_force_y);
    // std::vector<double> out_vertex_mass; P.vertex_mass.to_host(out_vertex_mass); write_vector(out_file, "vertex_mass", out_vertex_mass);
    H5Fclose(out_file);
    std::cout << "done" << std::endl;
}