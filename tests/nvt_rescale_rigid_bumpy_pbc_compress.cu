#include "hip/hip_runtime.h"
#include "particles/rigid_bumpy.cuh"
#include "utils/h5_io.hpp"
#include "utils/device_fields.cuh"
#include "utils/cuda_utils.cuh"
#include "routines/minimizers.cuh"
#include "integrators/velocity_verlet.cuh"
#include "integrators/damped_velocity_verlet.cuh"
#include "utils/output_manager.hpp"

// Run NVE dynamics with a rescaling thermostat
// For the first half of the run, incrementally (de)compress the system while maintaining the desired temperature
// The second half of the run is meant to equilibrate the system at the desired temperature

int main(int argc, char** argv) {
    if (argc != 7) {
        std::cerr << "Usage: " << argv[0] << " <in_path> <out_path> <n_steps> <phi_increment> <temperature_target> <dt_scale>" << std::endl;
        return 1;
    }
    std::string in_path = argv[1];
    std::string out_path = argv[2];
    const int n_steps = std::stoi(argv[3]);
    const double phi_increment = std::stod(argv[4]);  // total amount we will be (de)compressing the system
    const double temperature_target = std::stod(argv[5]);  // target temperature for the system
    const double dt_scale = std::stod(argv[6]);

    const int compression_frequency = std::min(n_steps, 100);  // how often we will be (de)compressing the system
    const double phi_step = phi_increment / (n_steps / compression_frequency);  // increment we will be (de)compressing the system by
    const int temperature_frequency = std::min(n_steps, 10);  // how often we will be setting the temperature of the system

    md::rigid_bumpy::RigidBumpy P;
    P.load_from_hdf5(in_path, "init");
    
    df::DeviceField1D<double> dt; dt.resize(P.n_systems()); dt.fill(dt_scale);
    md::integrators::VelocityVerlet vv(P, dt);
    vv.init();

    io::OutputManager<md::rigid_bumpy::RigidBumpy> om(P, out_path, 10, false);
    om.set_extra_init_fields({"packing_fraction", "box_size", "pos", "vel", "angle"});
    om.set_extra_final_fields({"packing_fraction", "box_size", "pos", "vel", "angle"});
    om.initialize();

    std::cout << "Running for " << n_steps << " steps" << std::endl;
    int i = 0;
    while (i < n_steps) {
        vv.step();
        om.step(i);
        if (i % 10000 == 0) {
            std::cout << "Step " << i << std::endl;
        }
        if (i % compression_frequency == 0) {
            P.increment_packing_fraction(phi_step);
        }
        if (i % temperature_frequency == 0) {
            P.set_temperature(temperature_target);
        }
        i++;
    }
    while (i < 2 * n_steps) {
        vv.step();
        om.step(i);
        if (i % 10000 == 0) {
            std::cout << "Step " << i << std::endl;
        }
        if (i % temperature_frequency == 0) {
            P.set_temperature(temperature_target);
        }
        i++;
    }
    om.finalize();
    std::cout << "Done" << std::endl;
}